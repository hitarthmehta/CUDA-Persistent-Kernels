#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <string.h>
#include "image.h"

#include "hip/hip_runtime.h"
#include ""

#define DEBUG

#define BLOCKSIZE 960
#define WIDTH 1920
#define HEIGHT 1080

#define BufferNum 2
#define StreamNum 3

#define VALID   1
#define INVALID 0

typedef struct rowFlag
{
  unsigned short flag;
  int row_data[WIDTH];
}rowFlag;

hipStream_t stream[StreamNum];
volatile int* done1;
volatile int* done2;
volatile int* processDone;

hipError_t checkCuda(hipError_t result)
{
    if(result != hipSuccess)
    {
        fprintf(stderr,"CUDA Runtime Error: %s\n",hipGetErrorString(result));
    }
return result;
}

void computeCPU(int * src, int * des, int size)
{
    int i;
    for(i = 0; i < size; i++)
    {
          des[i] = src[i] + 155;
    }
}
  
int verifyResults(int * d_result, int* cpu_result, int size)
{
  int i, er = 1;
  for (i = 0; i < size; i++)
  {
    if (d_result[i] != cpu_result[i])
    {
      er = 0;
      break;
    }
  }
  return er;
}


void printArray(int * a, int size)
{
  int i;
  for (i = 0; i < size; i++)
    printf("i: %d val: %d\n", i, a[i]);
}


void initArray(int * a, int size)
{
  int i;
  for (i = 0; i < size; i++)
    a[i] = 0;
}

__global__ void
ComputeKernel(struct rowFlag *devInPtr,int *devOutPtr,volatile int *processDone,volatile int * done1,volatile int * done2)
{
  // poll on received flag
  // if it is 1, then the previous row is received and process on the row
  // copy that row in output buffer and set done (mapped memory)
  int id1 = blockIdx.x * blockDim.x + threadIdx.x;
  int id2 = id1 + blockDim.x;
  
  unsigned short receivedFlag1,receivedFlag2;
  
  while(*processDone != 1)
  {
      receivedFlag1 = devInPtr[1].flag;
      receivedFlag2 = devInPtr[0].flag;
           
      if(receivedFlag1 == 1)
      {
          devOutPtr[(0 * WIDTH) + id1] = devInPtr[0].row_data[id1] + 155;
          devOutPtr[(0 * WIDTH) + id2] = devInPtr[0].row_data[id2] + 155;
          __syncthreads();
          if(id1 == 0)
          {  
            *done1 = VALID;
            receivedFlag1 = INVALID;
          }  
      }
      
      // set the received flag zero
      

      if(receivedFlag2 == 1 )
      {
          devOutPtr[(1 * WIDTH) + id1] = devInPtr[1].row_data[id1] + 155;
          devOutPtr[(1 * WIDTH) + id2] = devInPtr[1].row_data[id2] + 155;
          __syncthreads();
      
          // set the received flag zero
          if(id1 == 0)
          {  
            *done2 = VALID;
            receivedFlag2 = INVALID;
          }
      }
      
  }
}

__global__ void
StopKernel(volatile int *d_processDone)
{
    *d_processDone = 1;
}


int main()
{
//send the row with received flag of previous row
//poll on the received flag on GPU
//As soon as the received flag is 1,process the image and put it in another buffer and send back to CPU
//On CPU,set the done flag which is mapped memory,
  image *input,*output;
  int *h_a,*h_b,*cpu_b;
  int didIsendRow1=0,didIsendRow2=0;
  int numRowR=0,numRowS=0;
  int /**d_a,*/*d_b;
  rowFlag *d_a;
  int i,j;
  
  int rowSize = WIDTH;
  int imageSize = WIDTH * HEIGHT;
  int bufferSize = BufferNum * WIDTH;
  size_t rowByteSize = rowSize * sizeof(int);
  size_t imageByteSize = imageSize * sizeof(int);
  size_t bufferByteSize = bufferSize * sizeof(int);
  
  for(i = 0; i < StreamNum; i++)
    {
        checkCuda(hipStreamCreateWithFlags(&stream[i],hipStreamNonBlocking));
    }
  
  checkCuda(hipHostMalloc((void**)&h_a,imageByteSize));
  checkCuda(hipHostMalloc((void**)&h_b,imageByteSize));
  cpu_b = (int*)malloc(imageByteSize);
  
  checkCuda(hipMalloc((void**)&d_a,BufferNum * sizeof(rowFlag)));
  checkCuda(hipMalloc((void**)&d_b,bufferByteSize));
  checkCuda(hipMalloc((void**)&processDone,sizeof(int)));
  checkCuda(hipHostAlloc((void**)&done1,sizeof(int),hipHostMallocMapped));
  checkCuda(hipHostAlloc((void**)&done2,sizeof(int),hipHostMallocMapped));
  checkCuda(hipMemset((void*)processDone,0,sizeof(int)));
  checkCuda(hipMemset((void*)done1,0,sizeof(int)));
  checkCuda(hipMemset((void*)done2,0,sizeof(int)));
  initArray(h_a,imageSize);
  initArray(h_b,imageSize); 
  initArray(cpu_b,imageSize);
  
  #ifdef DEBUG
  printf("device buffers allocated\n");
  #endif
  
  ComputeKernel<<<1,BLOCKSIZE,0,stream[1]>>>(d_a,d_b,processDone,done1,done2);
  checkCuda(hipMemcpyAsync(d_a,h_a,sizeof(rowFlag)*2,hipMemcpyHostToDevice,stream[0]));
        
  #ifdef DEBUG
  printf("ComputeKernel launched\n");
  #endif
  rowFlag *temp;
  temp = (rowFlag*)malloc(sizeof(rowFlag));
        
  while(numRowR < HEIGHT)
  {
      //printf("numRowR: %d, numRowS: %d\n",numRowR,numRowS);
      //printf("d1: %d, d2: %d, s1: %d, s2: %d\n",*done1,*done2,didIsendRow1,didIsendRow2);
      if(*done1 == 1&&numRowR<HEIGHT)
      {
        //get(1);
        //printf("get1\n");
        checkCuda(hipMemcpyAsync(h_b+numRowR*WIDTH,d_b,rowByteSize,hipMemcpyDeviceToHost,stream[0]));
        numRowR++;
        didIsendRow1=0;
        *done1 = 0;
      }
      if(!didIsendRow1&&!(*done1)&&numRowS<HEIGHT)
      {
        //send row1
        //printf("send1\n");
        temp->flag=didIsendRow2;
        memcpy(temp->row_data,h_a+numRowS*WIDTH,rowByteSize);
        checkCuda(hipMemcpyAsync(d_a,temp,sizeof(temp),hipMemcpyHostToDevice,stream[0]));
        didIsendRow1=1;
        numRowS++;
        
      }
  
      
      if(*done2 == 1&&numRowR<HEIGHT)
      {
        //get(2);
        //printf("get2\n");
        checkCuda(hipMemcpyAsync(h_b+numRowR*WIDTH,d_b+WIDTH,rowByteSize,hipMemcpyDeviceToHost,stream[0]));
        numRowR++;
        *done2 = 0;
        didIsendRow2=0;
      }
  
      if(!didIsendRow2&&!(*done2)&&numRowS<HEIGHT)
      {
        //send row2
        //printf("send2\n");
        temp->flag=didIsendRow1;
        memcpy(temp->row_data,h_a+numRowS*WIDTH,rowByteSize);
        rowFlag * faltu=d_a+1;
        checkCuda(hipMemcpyAsync(faltu,temp,sizeof(temp),hipMemcpyHostToDevice,stream[0]));
        didIsendRow2=1;
        numRowS++;
      }
  }
  free(temp);
  StopKernel<<<1,1,0,stream[2]>>>(processDone);
  
  #ifdef DEBUG
  printf("StopKernel allocated\n");
  #endif
   
  computeCPU(h_a,cpu_b,imageSize);
  
  if(verifyResults(h_b,cpu_b,imageSize) == 1)
          printf("CPU Result and GPU  Result same;!!!\n");
      else
          printf("CPU Result and GPU  Result not same;!!!\n");
  
  input = loadImage("red.pgm");
  char newName[40] = "out";
  strcat(newName,input->name);
  output = createImage(newName,input->format,input->h,input->w,input->maxV);
  long x = 0;
  
  for(i = 0; i < input->h; i++)
  {
      for(j = 0; j < input->w; j++)
      {
          setPixelBW(input,i,j,h_a[x]);
          setPixelBW(output,i,j,h_b[x]);
          x++;
      }
  }
  
  saveImage(input);
  saveImage(output);
  deleteImage(input);
  deleteImage(output);
   return 0;
}